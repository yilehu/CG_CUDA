
#include <hip/hip_runtime.h>
__global__ void InitializeArray_GPU(double *Array,int Dim,double InitialValue)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while(tid<Dim)
	{
		Array[tid] = InitialValue;
		tid += gridDim.x*blockDim.x;
	}
}